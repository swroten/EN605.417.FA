#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "math_functions.h"
#include ""

#include <iostream>
#include <vector>
#include <sstream>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "hipblas.h"
#include "hipsolver.h"
#include "helper_cusolver.h"

using namespace std;

void PrintDeviceProperties()
{
	// Initialize Variables
	int device = 0;
	hipDeviceProp_t prop;

	// Get Properties of this device
	hipGetDeviceProperties(&prop, device);

	// Print Properties of this device
	printf("\n");
	printf("Device Number: %d\n", device);
	printf("  Device name: %s\n", prop.name);
	printf("  Warp Size: %i\n", prop.warpSize);
	printf("  Max Threads Per Block: %i\n", prop.maxThreadsPerBlock);
}

// Function to retrieving time
__host__ hipEvent_t get_time(void)
{
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

__global__ void init(unsigned int seed, hiprandState_t* states, const int numberOfElements)
{
	// Initialize Variables
	const int currentThreadIndex = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Exit if Thread out of bounds
	if (currentThreadIndex > numberOfElements) { return; }

	// Initialize Random Value
	hiprand_init(seed, currentThreadIndex, 0, &states[currentThreadIndex]);
}

__global__ void randoms(hiprandState_t* states, double* matrix, const int numberOfElements)
{
	// Initialize Variables
	const int currentThreadIndex = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Exit if Thread out of bounds
	if (currentThreadIndex > numberOfElements) { return; }

	// Set Random Number in Thread Index - make sure non-zero to prevent singular matrix for testing
	matrix[currentThreadIndex] = max(hiprand(&states[currentThreadIndex]) % 100, 1);
}

void GetRandomNumbersForMatrix(double *cpuMatrix, const int numberOfElements)
{
	// Initialize Variables
	int device = 0;
	int numberOfBlocks;
	hipDeviceProp_t prop;
	int numberOfThreads;
	double *gpuMatrix = 0;
	hiprandState_t* states;
	int maxThreadsPerBlock;
	const int numberOfBytesInMatrix = numberOfElements * sizeof(double);

	// Get Properties of this device
	hipGetDeviceProperties(&prop, device);

	// Get Max Threads Per Block
	maxThreadsPerBlock = prop.maxThreadsPerBlock;

	// Verify that Machine has GPU Installed by 
	//  selecting first GPU available.
	hipSetDevice(0);

	// Get Number of Blocks Required and Number of Threads
	numberOfBlocks = (int)(numberOfElements / maxThreadsPerBlock) + 1;
	numberOfThreads = (int)(numberOfElements % maxThreadsPerBlock);

	// Allocate GPU Memory for States
	hipMalloc((void**)&states, numberOfElements * sizeof(hiprandState_t));

	// Run Initialization
	init << <numberOfBlocks, numberOfThreads >> >((unsigned int)time(0), states, numberOfElements);

	// Allocate GPU Memory for input matrix
	hipMalloc((void**)&gpuMatrix, numberOfBytesInMatrix);

	// Add Random Numbers to Matrix
	randoms << <numberOfBlocks, numberOfThreads >> >(states, gpuMatrix, numberOfElements);

	// Copy Matrix Data From CPU Memory to GPU Memory
	hipMemcpy(cpuMatrix, gpuMatrix, numberOfBytesInMatrix, hipMemcpyDeviceToHost);

	// Free Allocated Memory
	hipFree(gpuMatrix);
}

std::string GetMatrixAsString(double *matrixElementsPntr, int squareMatrixDimension)
{
	// Initialize Variable
	std::ostringstream matrixAsStringStream;

	matrixAsStringStream << "{" << std::endl;

	// Step through each row in matrix
	for (int i = 0; i < squareMatrixDimension; i++)
	{
		// Spacing for initial elements
		matrixAsStringStream << "  ";

		// Step through each column in this row
		for (int j = 0; j < squareMatrixDimension; j++)
		{
			matrixAsStringStream << matrixElementsPntr[((i * squareMatrixDimension) + j)] << " ";
		}

		matrixAsStringStream << std::endl;
	}

	matrixAsStringStream << "};" << std::endl;

	// Return Matrix as String
	return matrixAsStringStream.str();
}

float GetInvertedMatrixCPU(double *cpuInvertedMatrix, const double *cpuLUMatrix, const int *cpuPivotMatrix, const int squareMatrixDimension)
{
	// Initialize Variables
	hipEvent_t stop;
	hipEvent_t start;
	double sumLowerTriangle = 0;
	double sumUpperTriangle = 0;
	float timeToCompleteInMs = 0;
	vector<double> solveArray(squareMatrixDimension, 0.0);
	vector<double> fowardSubstitutionArray(squareMatrixDimension, 0.0);
	vector<double> backwardSubtitutionArray(squareMatrixDimension, 0.0);

	// Keep Track of Start Time
	start = get_time();

	// Solve for the Identity Matrix using resuls of LU Decomposition
	//  Step through each row and solve
	for (int overallRowIndex = 0; overallRowIndex < squareMatrixDimension; overallRowIndex++)
	{
		// Initialize
		solveArray = vector<double>(squareMatrixDimension, 0.0);

		// Set to Identity
		solveArray[overallRowIndex] = 1;

		// Solve by doing foward substition
		for (int rowIndex = 0; rowIndex < squareMatrixDimension; rowIndex++)
		{
			// Set Accumulating sum to 0
			sumLowerTriangle = 0;

			// Step through Each Column
			for (int columnIndex = 0; columnIndex < rowIndex; columnIndex++)
			{
				// Accumulate Lower Triangle Sum
				sumLowerTriangle += (cpuLUMatrix[(rowIndex * squareMatrixDimension) + columnIndex] * fowardSubstitutionArray[columnIndex]);
			}

			// Perform Foward Substituition using Pivot Array and Accumulating Lower Triangle Sum
			fowardSubstitutionArray[rowIndex] = solveArray[cpuPivotMatrix[rowIndex]] - sumLowerTriangle;
		}

		// Solve by doing backward substition
		for (int rowIndex = squareMatrixDimension - 1; rowIndex >= 0; rowIndex--)
		{
			// Set Accumulating sum to 0
			sumUpperTriangle = 0;

			// Step through Each Column
			for (int columnIndex = rowIndex + 1; columnIndex < squareMatrixDimension; columnIndex++)
			{
				sumUpperTriangle += (cpuLUMatrix[(rowIndex * squareMatrixDimension) + columnIndex] * backwardSubtitutionArray[columnIndex]);
			}

			backwardSubtitutionArray[rowIndex] = ((fowardSubstitutionArray[rowIndex] - sumUpperTriangle) /
				cpuLUMatrix[(rowIndex * squareMatrixDimension) + rowIndex]);
		}

		// Perform final update to get Inverted Matrix
		for (int overallColumnIndex = 0; overallColumnIndex < squareMatrixDimension; overallColumnIndex++)
		{
			// Update Inverse Matrix
			cpuInvertedMatrix[(overallColumnIndex * squareMatrixDimension) + overallRowIndex] = backwardSubtitutionArray[overallColumnIndex];
		}
	}

	// Keep Track of Stop Time 
	stop = get_time();

	// Synchronize Events
	timeToCompleteInMs = 0;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timeToCompleteInMs, start, stop);

	// Return time required to complete
	return timeToCompleteInMs;
}

float GetLUDecompositionMatrixCPU(double *cpuInvertedMatrix, int *cpuPivotMatrix, const double *cpuMatrix, const int numberOfElements, const int squareMatrixDimension)
{
	// Initialize Variables
	hipEvent_t stop;
	hipEvent_t start;
	int maxValueIndex = 0;
	double largestValue = 0.0;
	float timeToCompleteInMs = 0;
	double matrixCurrentColumnValue = 0;
	double matrixLargestColumnValue = 0;
	int pivotMatrixCurrentColumnValue = 0;
	int pivotMatrixLargestColumnValue = 0;
	int *cpuMaxValueIndex = (int *)malloc(sizeof(int));

	// Copy Initial Matrix into Inverted Matrix
	hipMemcpy(cpuInvertedMatrix, cpuMatrix, numberOfElements * sizeof(double), hipMemcpyHostToHost);

	// Keep Track of Start Time
	start = get_time();

	// for each column in matrix
	for (int columnIndexInMatrix = 0; columnIndexInMatrix < squareMatrixDimension; columnIndexInMatrix++)
	{
		// Get all row elements in current column 
		largestValue = 0;
		for (int rowIndex = columnIndexInMatrix; rowIndex < squareMatrixDimension; rowIndex++)
		{
			// Get Max Row Value in Column
			if (abs(cpuInvertedMatrix[(rowIndex * squareMatrixDimension) + columnIndexInMatrix]) > largestValue)
			{
				largestValue = abs(cpuInvertedMatrix[(rowIndex * squareMatrixDimension) + columnIndexInMatrix]);
				maxValueIndex = rowIndex;
			}
		}

		// Update Pivot Matrix Indices and Values
		pivotMatrixCurrentColumnValue = cpuPivotMatrix[columnIndexInMatrix];
		pivotMatrixLargestColumnValue = cpuPivotMatrix[maxValueIndex];
		cpuPivotMatrix[columnIndexInMatrix] = pivotMatrixLargestColumnValue;
		cpuPivotMatrix[maxValueIndex] = pivotMatrixCurrentColumnValue;

		// Each Column Will Exchange Current Column Row Element with Pivot Element
		for (int columnIndex = 0; columnIndex < squareMatrixDimension; columnIndex++)
		{
			matrixCurrentColumnValue = cpuInvertedMatrix[(columnIndexInMatrix * squareMatrixDimension) + columnIndex];
			matrixLargestColumnValue = cpuInvertedMatrix[(maxValueIndex * squareMatrixDimension) + columnIndex];
			cpuInvertedMatrix[(columnIndexInMatrix * squareMatrixDimension) + columnIndex] = matrixLargestColumnValue;
			cpuInvertedMatrix[(maxValueIndex * squareMatrixDimension) + columnIndex] = matrixCurrentColumnValue;
		}

		// Perform Shurs Complement
		for (int rowIndex = columnIndexInMatrix + 1; rowIndex < squareMatrixDimension; rowIndex++)
		{
			cpuInvertedMatrix[(rowIndex * squareMatrixDimension) + columnIndexInMatrix] /= cpuInvertedMatrix[(columnIndexInMatrix * squareMatrixDimension) + columnIndexInMatrix];

			for (int columnIndex = columnIndexInMatrix + 1; columnIndex < squareMatrixDimension; columnIndex++)
			{
				cpuInvertedMatrix[(rowIndex * squareMatrixDimension) + columnIndex] -= (cpuInvertedMatrix[(rowIndex * squareMatrixDimension) + columnIndexInMatrix] *
					cpuInvertedMatrix[(columnIndexInMatrix * squareMatrixDimension) + columnIndex]);
			}
		}
	}

	// Keep Track of Stop Time 
	stop = get_time();

	// Synchronize Events
	timeToCompleteInMs = 0;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timeToCompleteInMs, start, stop);

	// Return time required to complete
	return timeToCompleteInMs;
}

float GetCuSparseInvertedMatrixGPU(double *cpuInvertedMatrix, const double *cpuMatrix, const int squareMatrixDimension)
{
	// Initialize Variables
	int batch = 1;
	int *info = NULL;
	hipEvent_t stop;
	hipEvent_t start;
	hipblasHandle_t handle;
	double *gpuLUDecompositionMatrix = NULL;
	int *gpuPivotMatrix = NULL;
	double *gpuInvertedMatrix = NULL;
	float timeToCompleteInMs = 0;

	// Allocate Device Memory
	hipMalloc((void **)&gpuPivotMatrix, sizeof(int)*squareMatrixDimension);
	hipMalloc((void **)&gpuLUDecompositionMatrix, sizeof(double)*squareMatrixDimension*squareMatrixDimension);
	hipMalloc((void **)&gpuInvertedMatrix, sizeof(double)*squareMatrixDimension*squareMatrixDimension);

	// Copy Data from CPU to GPU
	hipMemcpy(gpuLUDecompositionMatrix, cpuMatrix, sizeof(double)*squareMatrixDimension*squareMatrixDimension, hipMemcpyHostToDevice);
	hipMemcpy(gpuInvertedMatrix, cpuInvertedMatrix, sizeof(double)*squareMatrixDimension*squareMatrixDimension, hipMemcpyHostToDevice);

	// Initialize More Variables
	double **gpuInvertedMatrixArrayOfPointers = NULL;
	double **gpuLUDecompositionMatrixArrayOfPointers = NULL;
	double *cpuInvertedMatrixArray[] = { gpuInvertedMatrix };
	double *cpuLUDecompositionMatrixArray[] = { gpuLUDecompositionMatrix };

	// Create Handle
	hipblasCreate(&handle);

	// Allocate Memory to device arrays
	hipMalloc((void **)&gpuInvertedMatrixArrayOfPointers, sizeof(cpuInvertedMatrixArray));
	hipMalloc((void **)&gpuLUDecompositionMatrixArrayOfPointers, sizeof(cpuLUDecompositionMatrixArray));

	// Copy Data from CPU to GPU
	hipMemcpy(gpuInvertedMatrixArrayOfPointers, cpuInvertedMatrixArray, sizeof(cpuInvertedMatrixArray), hipMemcpyHostToDevice);
	hipMemcpy(gpuLUDecompositionMatrixArrayOfPointers, cpuLUDecompositionMatrixArray, sizeof(cpuLUDecompositionMatrixArray), hipMemcpyHostToDevice);

	// Keep Track of Start Time
	start = get_time();

	// Create Buffer
	hipMalloc((void **)&info, sizeof(int));

	// Initiailize Memory for Info
	hipMemset(info, 0, sizeof(int));

	// Perform LU Decomposition
	hipblasDgetrfBatched(handle, squareMatrixDimension, gpuLUDecompositionMatrixArrayOfPointers, squareMatrixDimension, gpuPivotMatrix, info, batch);

	// Compute Matrix Inverse
	hipblasDgetriBatched(handle, squareMatrixDimension, (const double **)gpuLUDecompositionMatrixArrayOfPointers, squareMatrixDimension, gpuPivotMatrix, gpuInvertedMatrixArrayOfPointers, squareMatrixDimension, info, batch);
	hipDeviceSynchronize();

	// Copy results from GPU Memory to Host Memory
	hipMemcpy(cpuInvertedMatrix, gpuInvertedMatrix, sizeof(double)*squareMatrixDimension*squareMatrixDimension, hipMemcpyDeviceToHost);

	// Keep Track of Stop Time 
	stop = get_time();

	// Synchronize Events
	timeToCompleteInMs = 0;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timeToCompleteInMs, start, stop);

	// Free up allocated memory
	if (handle) { hipblasDestroy(handle); }
	if (gpuPivotMatrix) { hipFree(gpuPivotMatrix); }
	if (gpuInvertedMatrix) { hipFree(gpuInvertedMatrix); }
	if (gpuInvertedMatrixArrayOfPointers) { hipFree(gpuInvertedMatrixArrayOfPointers); }
	if (gpuLUDecompositionMatrixArrayOfPointers) { hipFree(gpuLUDecompositionMatrixArrayOfPointers); }

	// return time required to complete matrix inversion
	return timeToCompleteInMs;
}

float InvertCPU(double *cpuInvertedMatrix, const double *cpuMatrix, const int squareMatrixDimension)
{
	// Initialize Variables
	float timeToGetLUDecompositionMatrix;
	float timeToInvertMatrixFromLUDecompositionAndPivotMatrix;
	const int numberOfElements = squareMatrixDimension * squareMatrixDimension;
	int *cpuPivotMatrixElementsPntr = (int *)malloc(squareMatrixDimension * sizeof(int));
	double *cpuLUMatrixElementsPntr = (double *)malloc(squareMatrixDimension * squareMatrixDimension * sizeof(double));

	// Initialize Pivot Matrix
	for (int i = 0; i < squareMatrixDimension; i++)
	{
		cpuPivotMatrixElementsPntr[i] = i;
	}

	// Add elements to matrix
	for (int i = 0; i < numberOfElements; i++)
	{
		cpuInvertedMatrix[i] = cpuMatrix[i];
		cpuLUMatrixElementsPntr[i] = cpuMatrix[i];
	}

	// On the CPU - Perform LU Decomposition to get LU Matrix and Pivot Matrix - returns time required to complete in ms
	timeToGetLUDecompositionMatrix = GetLUDecompositionMatrixCPU(cpuLUMatrixElementsPntr,
		cpuPivotMatrixElementsPntr,
		cpuMatrix,
		numberOfElements,
		squareMatrixDimension);

	// On the CPU - Use the LU Matrix and Pivot Matrix to get Inverte Matrix - returns time required to complete in ms 
	timeToInvertMatrixFromLUDecompositionAndPivotMatrix = GetInvertedMatrixCPU(cpuInvertedMatrix,
		cpuLUMatrixElementsPntr,
		cpuPivotMatrixElementsPntr,
		squareMatrixDimension);

	// Accumulate all Time Required to invert Matrix on cpu
	return (timeToGetLUDecompositionMatrix + timeToInvertMatrixFromLUDecompositionAndPivotMatrix);
}

float InvertGPU(double *cpuInvertedMatrix, const double *cpuMatrix, const int squareMatrixDimension)
{
	return GetCuSparseInvertedMatrixGPU(cpuInvertedMatrix, cpuMatrix, squareMatrixDimension);
}

// Main Function
int main(int argc, char *argv[])
{
	// Print Arguments for Debugging 
	std::cout << "Number of Arguments: " << argc << endl;
	std::cout << endl;

	// First Arg is Binary Name
	std::cout << "Binary Name: " << argv[0] << endl;
	std::cout << endl;

	// Second Arg is # of Threads 
	std::cout << "Matrix Dimension: " << argv[1] << endl;
	std::cout << endl;

	// Initialize Variables
	std::string userInput{ "" };
	bool invertSuccess = false;
	float cpuTimeToCompleteInMs = 0;
	float gpuTimeToCompleteInMs = 0;
	int numberOfRows = atoi(argv[1]);
	int numberOfColumns = atoi(argv[1]);
	double *cpuMatrixElementsPntr = 0;
	std::string cpuMatrixInversionResult{ "" };
	std::string gpuMatrixInversionResult{ "" };
	int numberOfElements = numberOfRows * numberOfColumns;
	double *cpuInvertedMatrixElementsPntrFromCPUComputation = 0;
	double *cpuInvertedMatrixElementsPntrFromGPUComputation = 0;
	int squareMatrixDimension = min(numberOfRows, numberOfColumns);

	// Allocate Memory
	cpuMatrixElementsPntr = (double *)malloc(numberOfElements * sizeof(double));
	cpuInvertedMatrixElementsPntrFromGPUComputation = (double *)malloc(numberOfElements * sizeof(double));
	cpuInvertedMatrixElementsPntrFromCPUComputation = (double *)malloc(numberOfElements * sizeof(double));

	// Get Random Values for Elements
	GetRandomNumbersForMatrix(cpuMatrixElementsPntr, numberOfElements);

	// Print Matrix as String
	std::cout << "Original Matrix:" << endl;
	std::cout << GetMatrixAsString(cpuMatrixElementsPntr, squareMatrixDimension) << endl;
	std::cout << endl;

	// Perform GPU Matrix Inversion 
	gpuTimeToCompleteInMs = InvertGPU(cpuInvertedMatrixElementsPntrFromGPUComputation, cpuMatrixElementsPntr, squareMatrixDimension);

	// Get GPU Computed Matrix Inversion as String
	gpuMatrixInversionResult = GetMatrixAsString(cpuInvertedMatrixElementsPntrFromGPUComputation, squareMatrixDimension);

	// Print Inverted Matrix (GPU) as String
	std::cout << "Inverted Matrix (GPU):" << endl;
	std::cout << gpuMatrixInversionResult << endl;
	std::cout << endl;

	// Perform CPU Matrix Inversion
	cpuTimeToCompleteInMs = InvertCPU(cpuInvertedMatrixElementsPntrFromCPUComputation, cpuMatrixElementsPntr, squareMatrixDimension);

	// Get CPU Computed Matrix Inversion as String
	cpuMatrixInversionResult = GetMatrixAsString(cpuInvertedMatrixElementsPntrFromCPUComputation, squareMatrixDimension);

	// Print Inverted Matrix (CPU) as String
	std::cout << "Inverted Matrix (CPU):" << endl;
	std::cout << cpuMatrixInversionResult << endl;
	std::cout << endl;

	// Check Results for success
	invertSuccess = (cpuMatrixInversionResult == gpuMatrixInversionResult);

	// Print out Results
	std::cout << "Results for Dimension " << squareMatrixDimension << ":" << endl;
	std::cout << "  Invert Equivalent: " << ((invertSuccess == 1) ? "Success" : "Failed") << endl;
	std::cout << "  CPU Time (ms):     " << cpuTimeToCompleteInMs << endl;
	std::cout << "  GPU Time (ms):     " << gpuTimeToCompleteInMs << endl;
	std::cout << "  Fastest:           " << ((cpuTimeToCompleteInMs < gpuTimeToCompleteInMs) ? "CPU" : "GPU") << endl;
	std::cout << endl;

	// Wait for user to close application
	std::cout << "Press Any Button to Exit..." << endl;

	// Get User Input
	getline(cin, userInput);

	// return
	return EXIT_SUCCESS;
}
